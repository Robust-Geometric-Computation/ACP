#include "hip/hip_runtime.h"
#include "encasement_gpu.h"

vector<PTR<Object<PPoly2>> > loadCurves(const char* fname) {

	vector<PTR<Object<PPoly2>> > curves;

	ifstream inFile(fname);
	string line;

	Parameter::enable();

	while (getline(inFile, line)) {

		istringstream buf(line);
		istream_iterator<std::string> beg(buf), end;

		vector<Parameter> coef;
		vector<int> pow;

		for (; beg != end;) {
			double c = stod((*beg));
			beg++;
			int i = stoi((*beg));
			beg++;
			int j = stoi((*beg));
			beg++;

			coef.push_back(Parameter::input(c));
			pow.push_back(i);
			pow.push_back(j);

		}

		curves.push_back(new Ellipse(PPoly2(coef.size(), &coef[0], &pow[0])));

	}

	Parameter::disable();

	return curves;
}

thrust::host_vector<i_pair<Number> > get_regions_gpu(PTR<Object<PPoly2>> f_in,
		PTR<Object<PPoly2>> g_in, double xl, double yl, double xu, double yu, int sub, int iterations) {

	//two device vectors, swapping buffers
	//first has the m intervals
	//subdivide so that second has floor(N / m) boxes for each of the m intervals (give iterators so no dead threads)
	//run the copy_if on that vector and keep the output iterator to know how many
	//swap buffers

	// input size
	int N = sub * sub;

	// define some types
	typedef thrust::device_vector<i_pair<Number> > d_vector_interval;
	typedef thrust::device_vector<int> d_vector_int;

	typedef d_vector_interval::iterator d_vector_interval_it;
	typedef d_vector_int::iterator d_vector_int_it;

	PPoly2 fp = f_in->get();
	PPoly2 fpx = fp.derX();
	PPoly2 fpy = fp.derY();
	PPoly2 fpxx = fpx.derX();
	PPoly2 fpxy = fpx.derY();
	PPoly2 fpyy = fpy.derY();

	PPoly2 gp = g_in->get();
	PPoly2 gpx = gp.derX();
	PPoly2 gpy = gp.derY();
	PPoly2 gpxx = gpx.derX();
	PPoly2 gpxy = gpx.derY();
	PPoly2 gpyy = gpy.derY();

	interval_gpu<Number> fcoef[fp.nt];
	interval_gpu<Number> fxcoef[fp.nt];
	interval_gpu<Number> fycoef[fp.nt];
	interval_gpu<Number> fxxcoef[fp.nt];
	interval_gpu<Number> fxycoef[fp.nt];
	interval_gpu<Number> fyycoef[fp.nt];

	interval_gpu<Number> gcoef[fp.nt];
	interval_gpu<Number> gxcoef[fp.nt];
	interval_gpu<Number> gycoef[fp.nt];
	interval_gpu<Number> gxxcoef[fp.nt];
	interval_gpu<Number> gxycoef[fp.nt];
	interval_gpu<Number> gyycoef[fp.nt];

	for (int i = 0; i < fp.nt; i++) {
		fcoef[i] = interval_gpu<T>(T(fp.a[i].lb()), T(fp.a[i].ub()));
	}
	for (int i = 0; i < fpx.nt; i++) {
		fxcoef[i] = interval_gpu<T>(T(fpx.a[i].lb()), T(fpx.a[i].ub()));
	}
	for (int i = 0; i < fpy.nt; i++) {
		fycoef[i] = interval_gpu<T>(T(fpy.a[i].lb()), T(fpy.a[i].ub()));
	}
	for (int i = 0; i < fpxx.nt; i++) {
		fxxcoef[i] = interval_gpu<T>(T(fpxx.a[i].lb()), T(fpxx.a[i].ub()));
	}
	for (int i = 0; i < fpxy.nt; i++) {
		fxycoef[i] = interval_gpu<T>(T(fpxy.a[i].lb()), T(fpxy.a[i].ub()));
	}
	for (int i = 0; i < fpyy.nt; i++) {
		fyycoef[i] = interval_gpu<T>(T(fpyy.a[i].lb()), T(fpyy.a[i].ub()));
	}

	for (int i = 0; i < gp.nt; i++) {
		gcoef[i] = interval_gpu<T>(T(gp.a[i].lb()), T(gp.a[i].ub()));
	}
	for (int i = 0; i < gpx.nt; i++) {
		gxcoef[i] = interval_gpu<T>(T(gpx.a[i].lb()), T(gpx.a[i].ub()));
	}
	for (int i = 0; i < gpy.nt; i++) {
		gycoef[i] = interval_gpu<T>(T(gpy.a[i].lb()), T(gpy.a[i].ub()));
	}
	for (int i = 0; i < gpxx.nt; i++) {
		gxxcoef[i] = interval_gpu<T>(T(gpxx.a[i].lb()), T(gpxx.a[i].ub()));
	}
	for (int i = 0; i < gpxy.nt; i++) {
		gxycoef[i] = interval_gpu<T>(T(gpxy.a[i].lb()), T(gpxy.a[i].ub()));
	}
	for (int i = 0; i < gpyy.nt; i++) {
		gyycoef[i] = interval_gpu<T>(T(gpyy.a[i].lb()), T(gpyy.a[i].ub()));
	}

	cuda_poly2<Number> hf(fcoef, fp.m, fp.nt);
	cuda_poly2<Number> hfx(fxcoef, fpx.m, fpx.nt);
	cuda_poly2<Number> hfy(fycoef, fpy.m, fpy.nt);
	cuda_poly2<Number> hfxx(fxxcoef, fpxx.m, fpxx.nt);
	cuda_poly2<Number> hfxy(fxycoef, fpxy.m, fpxy.nt);
	cuda_poly2<Number> hfyy(fyycoef, fpyy.m, fpyy.nt);

	cuda_poly2<Number> hg(gcoef, gp.m, gp.nt);
	cuda_poly2<Number> hgx(gxcoef, gpx.m, gpx.nt);
	cuda_poly2<Number> hgy(gycoef, gpy.m, gpy.nt);
	cuda_poly2<Number> hgxx(gxxcoef, gpxx.m, gpxx.nt);
	cuda_poly2<Number> hgxy(gxycoef, gpxy.m, gpxy.nt);
	cuda_poly2<Number> hgyy(gyycoef, gpyy.m, gpyy.nt);

	cuda_poly2<Number> * f = hf.copy_device();
	cuda_poly2<Number> * fx = hfx.copy_device();
	cuda_poly2<Number> * fy = hfy.copy_device();
	cuda_poly2<Number> * fxx = hfxx.copy_device();
	cuda_poly2<Number> * fxy = hfxy.copy_device();
	cuda_poly2<Number> * fyy = hfyy.copy_device();

	cuda_poly2<Number> * g = hg.copy_device();
	cuda_poly2<Number> * gx = hgx.copy_device();
	cuda_poly2<Number> * gy = hgy.copy_device();
	cuda_poly2<Number> * gxx = hgxx.copy_device();
	cuda_poly2<Number> * gxy = hgxy.copy_device();
	cuda_poly2<Number> * gyy = hgyy.copy_device();

	// allocate storage for array
	d_vector_int values(N);

	// initialize array to [0, 1, 2, ... ]
	thrust::sequence(values.begin(), values.end());

	// allocate output storage, here we conservatively assume all values will be copied
	d_vector_interval sub_div_intervals(values.size());
	d_vector_interval predicate_intervals(values.size());

	d_vector_interval_it predicate_intervals_end = predicate_intervals.begin()
			+ 1;
	// setup initial interval
	i_pair<Number> initial;
	initial.x = interval_gpu<Number>(xl, xu);
	initial.y = interval_gpu<Number>(yl, yu);

	sub_div_intervals[0] = initial;
	predicate_intervals[0] = initial;

	int num_intervals = 1;

	for (int index = 0; index < iterations; index++) {

		// subdivide the interval into subintervals

		int subs_per_thread = N / num_intervals;

		//subs_per_thread = max(4, subs_per_thread);

		cout << "subs_per: " << subs_per_thread << endl;

		int rows = int(sqrtf(subs_per_thread));
		int cols = subs_per_thread / rows;

		cout << "rows: " << rows << endl;
		cout << "cols: " << cols << endl;

		subs_per_thread = rows * cols;

		cout << "subs_per: " << subs_per_thread << endl;

		int total_threads = subs_per_thread * num_intervals;

		cout << "total_threads: " << total_threads << endl;

		struct subdivide<Number> subdivide_functor(total_threads,
				thrust::raw_pointer_cast(&predicate_intervals[0]),
				num_intervals);

		thrust::transform(values.begin(), values.begin() + total_threads,
				sub_div_intervals.begin(), subdivide_functor);

		// copy all intervals that ARE ambiguous
		//struct is_ambiguous<Number> ambiguous_functor(f, g);
		struct is_ambiguous_quad<Number> ambiguous_functor(f, fx, fy, fxx, fxy,
				fyy, g, gx, gy, gxx, gxy, gyy);

		predicate_intervals_end = thrust::copy_if(sub_div_intervals.begin(),
				sub_div_intervals.begin() + total_threads,
				predicate_intervals.begin(), ambiguous_functor);

		num_intervals = int(
				predicate_intervals_end - predicate_intervals.begin());

		//FOR TESTING

		struct function_value<Number> function_value_functor(f, g);

		d_vector_interval function_values(num_intervals);

		thrust::transform(predicate_intervals.begin(), predicate_intervals_end,
				function_values.begin(), function_value_functor);

		thrust::host_vector<i_pair<Number> > hfunction_values = function_values;
		thrust::host_vector<i_pair<Number> > hintervals(
				predicate_intervals.begin(), predicate_intervals_end);

		printf("removed intervals size: %d\n", num_intervals);

		//for(int i = 0; i < hintervals.size(); i++) {
		//    printf("f([%f, %f][%f, %f]) -> [%f, %f]\n", hintervals[i].x.lower(), hintervals[i].x.upper(), hintervals[i].y.lower(), hintervals[i].y.upper(), hfunction_values[i].x.lower(), hfunction_values[i].x.upper());
		//    printf("g([%f, %f][%f, %f]) -> [%f, %f]\n\n", hintervals[i].x.lower(), hintervals[i].x.upper(), hintervals[i].y.lower(), hintervals[i].y.upper(), hfunction_values[i].y.lower(), hfunction_values[i].y.upper());
		//}

	}

	return thrust::host_vector < i_pair<Number>
			> (predicate_intervals.begin(), predicate_intervals_end);
}

vector<Rectangle *> get_all_rects(PTR<Object<PPoly2>> f, PTR<Object<PPoly2>> g, double xl,
		double yl, double xu, double yu, int sub, int iterations) {

	thrust::host_vector<i_pair<Number> > hintervals = get_regions_gpu(f, g, xl,
			yl, xu, yu, sub, iterations);

	vector<Rectangle *> all_rects;

	for (int i = 0; i < hintervals.size(); i++) {
		all_rects.push_back(
				new Rectangle(hintervals[i].x.lower(), hintervals[i].y.lower(),
						hintervals[i].x.upper(), hintervals[i].y.upper()));
	}

	return all_rects;

}
vector<Rectangle *> get_regions(PTR<Object<PPoly2>> f, PTR<Object<PPoly2>> g, double xl,
		double yl, double xu, double yu, int sub, int iterations) {

	hipGetLastError();
	hipDeviceReset();
	hipSetDevice(0);

	vector<Rectangle *> all_rects = get_all_rects(f, g, xl, yl, xu, yu, sub, iterations);

	vector<Rectangle *> bounding_boxes;

	map<int, set<Rectangle *> > m = connected_components(all_rects);

	for (map<int, set<Rectangle *> >::iterator it = m.begin(); it != m.end();
			it++) {

		set<Rectangle *> s = (*it).second;

		double rxl, ryl, rxu, ryu;
		bool first = true;

		for (set<Rectangle *>::iterator rectit = s.begin(); rectit != s.end();
				rectit++) {
			Rectangle * rect = *(rectit);
			if (first) {
				rxl = rect->box.x.lb();
				ryl = rect->box.y.lb();
				rxu = rect->box.x.ub();
				ryu = rect->box.y.ub();
				first = false;
			} else {
				rxl = fmin(rxl, rect->box.x.lb());
				ryl = fmin(ryl, rect->box.y.lb());
				rxu = fmax(rxu, rect->box.x.ub());
				ryu = fmax(ryu, rect->box.y.ub());
			}
		}

		bounding_boxes.push_back(new Rectangle(rxl, ryl, rxu, ryu));

	}

	return bounding_boxes;

}
